
#include "hip/hip_runtime.h"

#include "hiprand/hiprand.h"
#include <stdio.h>
#include <time.h>
#include <math.h>
#include <cstdio>

__global__ void addTen(float* d, int wielkosc){

	int ThreadsInBlock = blockDim.x * blockDim.y* blockDim.z;

	int PositionThreadInBlock = threadIdx.x + threadIdx.y* blockDim.x + threadIdx.z * blockDim.x*blockDim.y;

	int PositionBlockInGrid = blockIdx.x + blockIdx.y*gridDim.x;

	int index = PositionBlockInGrid * ThreadsInBlock + PositionThreadInBlock;

	if(index < wielkosc){
		d[index] += 10;
	}
}

int main()
{

  hiprandGenerator_t generator;
  hiprandCreateGenerator(&generator,HIPRAND_RNG_PSEUDO_MTGP32);
  hiprandSetPseudoRandomGeneratorSeed(generator,time(0));

   const int wielkosc = 123456;
   int rozmiar = sizeof(float)*wielkosc;
   float tablica[wielkosc];
   float *d;
   hipMalloc(&d,rozmiar);
   hiprandGenerateUniform(generator,d,wielkosc);

   dim3 block(8,8,8);
   dim3 grid(16,16);

   addTen<<<grid,block>>>(d,wielkosc);

   hipMemcpy(tablica,d,rozmiar,hipMemcpyDeviceToHost);

   hipFree(d);

   for (int i = 0; i < wielkosc; i++)
   {
	   printf("\n %f", tablica[i]);
   }

}